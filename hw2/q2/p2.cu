/*


Single Author info:
arajend4 Ayushi Rajendra Kumar

*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
//#include <cuda_runtime.h>
/* first grid point */
#define   XI              0.0
/* last grid point */
#define   XF              M_PI
typedef   double   FP_PREC;
/* function declarations */
double     fn(double);
void        print_function_data(int, double*, double*, double*);
int         main(int, char**); 
__global__ void calc_area(double*,double*,double*,double,int);
__device__ void fn1(double*, double*);
int main (int argc, char *argv[])
{
        int NGRID;
        if(argc > 1)
            NGRID = atoi(argv[1]);
        else 
        {
                printf("Please specify the number of grid points.\n");
                exit(0);
        }
        //loop index
        int         i;
        double  h, area;

        double *inf = (double *)malloc(sizeof(double) * (NGRID + 1) );
        double  *xc = (double *)malloc(sizeof(double)* (NGRID + 1));
        double  *yc = (double*)malloc(sizeof(double) * (NGRID + 1));
        double *inf_d;
	double *xc_d;
	double *yc_d;
	hipMalloc((void ** )&inf_d,sizeof(double)*(NGRID+1)); 
	hipMalloc((void ** )&xc_d,sizeof(double)*(NGRID+1)); 
	hipMalloc((void ** )&yc_d,sizeof(double)*(NGRID+1)); 
	int blockSize=4;
	int nBlocks = (NGRID)/blockSize + ((NGRID)%blockSize == 0?0:1);
        //construct grid
	for (i = 1; i <= NGRID ; i++)
        {
                xc[i] = XI + (XF - XI) * (double)(i - 1)/(double)(NGRID - 1);
        }
        
        int  imin, imax;  

        imin = 1;
        imax = NGRID;
        //define the function
      //  for( i = imin; i <= imax; i++ )
       // {
         //       yc[i] = fn(xc[i]);
       // }
	


        inf[0] = 0.0;
        h = (XF - XI) / (NGRID - 1);
        
	hipMemcpy(xc_d,&xc[1],sizeof(double)*(NGRID+1),hipMemcpyHostToDevice);
	hipMemcpy(inf_d,inf,sizeof(double)*(NGRID+1),hipMemcpyHostToDevice);
        calc_area<<<nBlocks,blockSize>>>(inf_d,xc_d,yc_d,h,NGRID);
	hipMemcpy(&inf[1],inf_d,sizeof(double)*(NGRID),hipMemcpyDeviceToHost);
	hipMemcpy(&yc[1],yc_d,sizeof(double)*(NGRID),hipMemcpyDeviceToHost);
        for(i = 1 ; i <= NGRID; ++i){
           //  x += h;
           //  y2 = fn(x);
         //   area = ( yc[i] + yc[i-1]) * h / 2;
            inf[i] = inf[i]+inf[i-1];
        }


        print_function_data(NGRID, &xc[1], &yc[1], &inf[1]);

	hipFree(inf_d);
	hipFree(xc_d);
	hipFree(yc_d);
        //free allocated memory 
        free(xc);
        free(yc);
        free(inf);

        return 0;
}

//prints out the function and its derivative to a file
void print_function_data(int np, double *x, double *y, double *dydx)
{
        int   i;

        char filename[1024];
        sprintf(filename, "fn-%d.dat", np);

        FILE *fp = fopen(filename, "w");

        for(i = 0; i < np; i++)
        {
                fprintf(fp, "%f %f %f\n", x[i], y[i], dydx[i]);
        }

        fclose(fp);
}


__global__ void calc_area(double *area,double *xc,double *yc,double h,int N)
{
	int idx=blockIdx.x*blockDim.x + threadIdx.x;
	//double y[2];
	

	fn1(&xc[idx],&yc[idx]);
	//fn1(&xc[idx-1],&y[1]);
	//printf("value of yc is idx %d %lf\n",idx,yc[idx]);
	__syncthreads();

	if(idx>0) area[idx]=(yc[idx]+yc[idx-1])*h/2;
//	printf("yc is %lf,h is %d,area is %lf\n",yc[idx],h,area[idx]);
}

__device__ void fn1(double* x,double* y)
{
  *y= *x * *x;
 //  printf("fn is %lf\n",*y);
}


